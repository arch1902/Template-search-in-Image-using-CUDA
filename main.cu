#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>

using namespace std;

// __global__ void add(int *a, int *b, int *c)
// {
//   *c = *a + *b;
// }

void input(int n, int m, string filename, float *arr, int *R, int *G, int *B, bool flag, int &avg)
{
  int r,g,b;
  int val;
  ifstream file(filename);
  for(int i=0;i<n;i++){
      for(int j=0;j<m;j++){
          file >> r >> g >> b;
          R[i*m + j] = r;
          G[i*m + j] = g;
          B[i*m + j] = b;

          //TODO check if conversion to float array is required or not
          arr[i*m + j] = (r+g+b)/3;
          val += arr[i*m+j];
      }
  }
  if(!flag) cout<<"Data Image read !"<<endl;
  file.close();
  if(flag) avg = val/(m*n);
}


// TODO optimize this function, add elements row-wise in parallel and then take their sum
// User thread shared memory


__device__
void computeRMSD(float *dataR, float *dataG, float *dataB, int *queryR, int *queryG, int *queryB, int n, int m, int query_n, int query_m, int x, int y, int &rmsd)
{
    float sum = 0;
    for(int i=0;i<query_n;i++){
        for(int j=0;j<query_m;j++){
        sum += pow((dataR[(x+i)*m + (y+j)] - queryR[i*query_m + j]),2);
        sum += pow((dataG[(x+i)*m + (y+j)] - queryG[i*query_m + j]),2);
        sum += pow((dataB[(x+i)*m + (y+j)] - queryB[i*query_m + j]),2);
        }
    }
    rmsd = sqrt(sum);
}

// TODO store R,G,B pointers in some array or in some sort of struct
__global__
void computeImageSummary(float *data, float *dataR, float *dataG, float *dataB, int *queryData, int *queryR, int *queryG, int *queryB, int n, int m, int query_n, int query_m, float *result, int QueryVal, int threshold)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx> m*n*3) return;
    int x = idx/(m*3);
    int y = (idx%(m*3))/3;
    int orientation = (idx%(m*3))%3;
    long long val = 0;
    //printf("x %d y %d or %d \n",x,y,orientation);
    
    // printf("Inside, 1.)%d\t2.)%d\t3.)%d\t4.)%d\t5.)Idx:%d\n",x,y,query_n,query_m,idx);
    // printf("N,M, 1.)%d\t2.)%d\n",n,m);
    // if(x+query_n>=n || y+query_m>=m)return;

    int xmin,xmax,ymin,ymax;

    if(orientation==0){
        xmin = 0;
        xmax = query_n;
        ymin = 0;
        ymax = query_m;
    }else if(orientation==1){ // +45 degrees
        ymin = -(query_n)/(1.414);
        ymax = query_m/(1.414);
        xmin = 0;
        xmax = (query_m + query_n)/(1.414);
    }else { // -45 degrees
        ymin = 0;
        ymax = (query_m + query_n)/(1.414);
        xmin = -(query_m)/(1.414);
        xmax = (query_n)/(1.414);
    }
    

    for(int i=xmin;i<xmax;i++){
        for(int j=ymin;j<ymax;j++){
            if(x+i >= n or x+i < 0 or y+j >= m or y+j < 0) val += 255;
            else val += data[(x+i)*m + (y+j)];
        }
    }

    int boxSize = (xmax-xmin)*(ymax-ymin);

    result[idx] = (float)(val)/boxSize;

    if(abs(result[idx] -QueryVal)<=threshold)
    {
        computeRMSD(dataR,dataG,dataB,queryR,queryG,queryB,n,m,query_n,query_m,x,y,rmsd);
        // computeRMSD(data,queryData,n,m,query_n,query_m,x,y);
    }

    // if(result[idx]>71.5 and result[idx]<72.5) {
    //     printf("%d %d %f \n",x,y,result[idx]);
    // }

    // if(x==290 and y==120 and orientation==1){
    //     printf("%.6f \n", result[idx]);
    // }

    // printf("Sub region (%d,%d) avg value: %d\n",x,y,idx,result[idx]);

    // print blockDim
    //printf("BlockDim: %d\t, BlockIdx: %d\t, ThreadIdx: %d\n",blockDim.x,blockIdx.x,threadIdx.x);
}

int main(int argc, char* argv[]){

    string data_image_path = argv[1];
    string query_image_path = argv[2];
    double threshold1 = stod(argv[3]);
    double threshold2 = stod(argv[4]);
    int n = stoi(argv[5]);
    int rows, cols;
    int imageSummaryQuery;

    // Read the data image
    ifstream data_image_file(data_image_path);
    data_image_file >> rows >> cols;
    data_image_file.close();
    float *data_imageV;
    int *data_imageR;
    int *data_imageG;
    int *data_imageB;
    hipMallocManaged(&data_imageV, rows*cols*sizeof(float));
    hipMallocManaged(&data_imageR, rows*cols*sizeof(int));
    hipMallocManaged(&data_imageG, rows*cols*sizeof(int));
    hipMallocManaged(&data_imageB, rows*cols*sizeof(int));

    input(rows,cols,data_image_path,data_imageV,data_imageR,data_imageG,data_imageB,false,imageSummaryQuery);

    // Read the query image
    ifstream query_image_file(query_image_path);
    int query_rows, query_cols;
    query_image_file >> query_rows >> query_cols;
    query_image_file.close();
    float *query_imageV;
    int *query_imageR;
    int *query_imageG;
    int *query_imageB;

    hipMallocManaged(&query_imageV, query_rows*query_cols*sizeof(float));
    hipMallocManaged(&query_imageR, query_rows*query_cols*sizeof(int));
    hipMallocManaged(&query_imageG, query_rows*query_cols*sizeof(int));
    hipMallocManaged(&query_imageB, query_rows*query_cols*sizeof(int));
    
    input(query_rows,query_cols,query_image_path,query_imageV,query_imageR,query_imageG,query_imageB,true,imageSummaryQuery);

    cout<<"Query Image Summary: "<<imageSummaryQuery<<endl;

    // Compute the image summary
    // int imageSummaryCuda[rows-query_rows][cols-query_cols];

    // int block_size = 256;
    // int grid_size = (rows-query_rows)/block_size + 1;

    int imageSummarySize = (cols)*(rows)*3;

    float *imageSummary;
    // hipMalloc((void **)&data_imageVCuda,sizeof(int)*rows*cols);
    // hipMalloc((void **)&imageSummaryCuda,sizeof(int)*imageSummarySize);
    
    // hipMemcpy(data_imageVCuda,&data_imageV,sizeof(int)*rows*cols,hipMemcpyHostToDevice);


    hipMallocManaged(&imageSummary, imageSummarySize*sizeof(float));

    int num_blocks = (rows*cols*3)/1024 + 1;

    // computeImageSummary<<<grid_size,block_size>>>(data_imageVCuda,rows,cols,query_rows,query_cols,imageSummaryCuda);

    computeImageSummary<<<num_blocks, 1024>>>(data_imageV,query_imageV,rows,cols,query_rows,query_cols,imageSummary,imageSummaryQuery,threshold1);

    hipError_t err = hipGetLastError();

     if ( err != hipSuccess )
     {
        printf("CUDA Error: %s\n", hipGetErrorString(err));       

        // Possibly: exit(-1) if program cannot continue....
     }


    hipDeviceSynchronize();

    // for(int i=0;i<imageSummarySize;i++){
    //     cout<<"x:"<<i/(cols*3)<<", y:"<<(i%(cols*3))/3<<", orientation:"<<(i%(cols*3))%3<<" -> "<<imageSummary[i]<<endl;
    // }


    return 0;

}