
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <algorithm>
#include <chrono>
#include <map>

using namespace std;
using namespace std::chrono;

void dataInput(int n, int m, string filename, float *arr, int *R, int *G, int *B, float *compressedData)
{
  int r,g,b;
  float val = 0;
  ifstream file(filename);
  file >> n >> m;

  for(int i=n-1;i>=0;i--)
  {
      float valRow = 0;
      for(int j=0;j<m;j++){
          file >> r >> g >> b;
          R[i*m + j] = r;
          G[i*m + j] = g;
          B[i*m + j] = b;

          //TODO check if conversion to float array is required or not
          arr[i*m + j] = (float)(r+g+b)/3.0;
          valRow += arr[i*m+j];
          compressedData[i*m + j] = valRow;
      }
      val += valRow;
  }
  cout<<"Data Image read !"<<endl;
  file.close();
}

void queryInput(int n, int m, string filename, float *arr, int *R, int *G, int *B, float &avg)
{
  int r,g,b;
  float val = 0;
  ifstream file(filename);
  file >> n >> m;

  for(int i=n-1;i>=0;i--){
      for(int j=0;j<m;j++){
          file >> r >> g >> b;
          R[i*m + j] = r;
          G[i*m + j] = g;
          B[i*m + j] = b;

          //TODO check if conversion to float array is required or not
          arr[i*m + j] = (float)(r+g+b)/3.0;
          val += arr[i*m+j];
      }
  }
  cout << "Query Image read!\n";
  file.close();
  avg = val/((float)m*(float)n);
}

// TODO optimize this function, add elements row-wise in parallel and then take their sum
// User thread shared memory

__device__
void bilinearInterpolation(int *data, float x, float y,int n, int m, float &val)
{
    
    float downx  = floor(x);
    float upx    = downx + 1.0;
    
    float lefty  = floor(y);
    float righty = lefty + 1.0;

    float topleft;
    float topright;
    float bottomleft;
    float bottomright;

    if(upx>=n or lefty<0){
        topleft = 0;
    }else {
        topleft = data[int(upx)*m+int(lefty)];
    }

    if(upx>=n or righty>=m){
        topright = 0;
    }else {
        topright = data[int(upx)*m+int(righty)];
    }

    if(downx<0 or lefty<0){
        bottomleft = 0;
    }else {
        bottomleft = data[int(downx)*m+int(lefty)];
    }

    if(downx<0 or righty>=m){
        bottomright = 0;
    }else {
        bottomright = data[int(downx)*m+int(righty)];
    }   

    // F(x,y) = z00*(1-x)*(1-y) + z10*x*(1-y) + z01*(1-x)*y + z11*x*y
    float f = bottomleft*(righty - y)*(upx - x) + topleft*(righty - y)*(x - downx) + bottomright*(y-lefty)*(upx - x) + topright*(y-lefty)*(x-downx);
    val = f;

    //printf("%f %f %f %f", topleft, bottomleft, topright, bottomright);
}

__device__
void computeRMSD(int *dataR, int *dataG, int *dataB, int *queryR, int *queryG, int *queryB, int n, int m, int query_n, int query_m, int idx, float &rmsd)
{
    int x = idx/(m*3);
    int y = (idx%(m*3))/3;
    // (x,y) is the bottom left pixel coordinate of the data image
    int orientation = (idx%(m*3))%3;
    double sum  = 0;
    // float temp = 0;

    float ptx,pty;

    for(int i=0;i<query_n;i++)
    {
        for(int j=0;j<query_m;j++)
        {
            int query_cord = i*query_m+j;
            if(orientation==0)
            {
                ptx = x+i;
                pty = y+j;

                if(ptx <0 or ptx>=n or pty <0 or pty>=m){
                    sum += (queryR[query_cord])*(queryR[query_cord]);
                    sum += (queryG[query_cord])*(queryG[query_cord]);
                    sum += (queryB[query_cord])*(queryB[query_cord]);
                    continue;
                }

                int data_cord = int(ptx)*m+int(pty);
                
                sum += (dataR[data_cord] - queryR[query_cord])*(dataR[data_cord] - queryR[query_cord]);
                sum += (dataG[data_cord] - queryG[query_cord])*(dataG[data_cord] - queryG[query_cord]);
                sum += (dataB[data_cord] - queryB[query_cord])*(dataB[data_cord] - queryB[query_cord]);
            }
            else if(orientation==1) // +45
            {
                ptx = (float)x + ((float)(i+j))/sqrt(2.0);
                pty = (float)y + ((float)(j-i))/sqrt(2.0);

                float r;bilinearInterpolation(dataR,ptx,pty,n,m,r);
                float g;bilinearInterpolation(dataG,ptx,pty,n,m,g);
                float b;bilinearInterpolation(dataB,ptx,pty,n,m,b);

                // if(i==query_n-10 and j==query_m-10) {
                //     printf("R:%f ",dataR[int(ptx)*m + int(pty)]);
                //     printf("ptx:%f, pty:%f, r:%f, g:%f, b:%f \n",ptx,pty,r,g,b);
                // }

                sum += (r - float(queryR[query_cord]))*(r - float(queryR[query_cord]));
                sum += (g - float(queryG[query_cord]))*(g - float(queryG[query_cord]));
                sum += (b - float(queryB[query_cord]))*(b - float(queryB[query_cord]));

            }
            else if(orientation==2)  // -45
            {
                ptx = (float)x + ((float)(i-j))/sqrt(2.0);
                pty = (float)y + ((float)(i+j))/sqrt(2.0);

                float r;bilinearInterpolation(dataR,ptx,pty,n,m,r);
                float g;bilinearInterpolation(dataG,ptx,pty,n,m,g);
                float b;bilinearInterpolation(dataB,ptx,pty,n,m,b);

                sum += (r - float(queryR[query_cord]))*(r - float(queryR[query_cord]));
                sum += (g - float(queryG[query_cord]))*(g - float(queryG[query_cord]));
                sum += (b - float(queryB[query_cord]))*(b - float(queryB[query_cord]));
            }
        }
    }
    rmsd = sqrt(sum/(query_n*query_m*3));
    //printf("x:%d, y:%d, ori:%d, sum:%lf \n",x,y,orientation,sum);

}

// TODO store R,G,B pointers in some array or in some sort of struct
__global__
void computeImageSummary(float *data, int *dataR, int *dataG, float *compressedData, int *dataB, float *queryData, int *queryR, int *queryG, int *queryB, int n, int m, int query_n, int query_m, float *result, float *rmsdValues, float QueryVal, float threshold)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx> m*n*3) return;
    int x = idx/(m*3);
    int y = (idx%(m*3))/3;
    int orientation = (idx%(m*3))%3;
    long long val = 0;
    //printf("x %d y %d or %d \n",x,y,orientation);

    int xmin,xmax,ymin,ymax;

    if(orientation==0){
        xmin = 0;
        xmax = query_n;
        ymin = 0;
        ymax = query_m;
    }else if(orientation==1){ // +45 degrees
        ymin = -floor((query_n)/(sqrt(2.0)));
        ymax = ceil(query_m/(sqrt(2.0)));
        xmin = 0;
        xmax = ceil((query_m + query_n)/(sqrt(2.0)));
    }else { // -45 degrees
        ymin = 0;
        ymax = (query_m + query_n)/(sqrt(2.0));
        xmin = -(query_m)/(sqrt(2.0));
        xmax = (query_n)/(sqrt(2.0));
    }
    

   if((x+xmin)<0 || (x+xmax)>=n || (y+ymin)<0 || (y+ymax)>=m)
    {
        result[idx] = -1;
        rmsdValues[idx] = -1;
        return;
    }

    for(int i=x+xmin;i<x+xmax;i++)
    {
        val += (compressedData[i*m + (y+ymax-1)] - compressedData[i*m + (y+ymin)] + data[i*m + y+ymin]);
    }

    // for(int i=xmin;i<xmax;i++){
    //     for(int j=ymin;j<ymax;j++){
    //         if(x+i >= n or x+i < 0 or y+j >= m or y+j < 0) val += 0;
    //         else val += data[(x+i)*m + (y+j)];
    //     }
    // }


    float boxSize = (xmax-xmin)*(ymax-ymin);

    result[idx] = (float)(val)/boxSize;

    float rmsd = -1;


    if(abs(result[idx] - QueryVal) <= threshold)
    {
        computeRMSD(dataR,dataG,dataB,queryR,queryG,queryB,n,m,query_n,query_m,idx,rmsd);
    }

    rmsdValues[idx] = rmsd;

    // if(result[idx]>195 and result[idx]<198) {
    //     printf("%d %d %f %f \n",x,y,result[idx],rmsdValues[idx]);
    // }

    // if(x==100 and y==4772 and orientation==2){
    //     printf("RMSD:%.6f AVG:%f %f %d \n", rmsdValues[idx], result[idx], threshold, QueryVal);
    // }
}

struct triplet
{
    int x,y;
    int orientation; // 0 for 0 degrees, 1 for 45 degrees, 2 for -45 degrees
    float val;
    float summary;
};

bool sortbyVal(const triplet &a, 
              const triplet &b) 
{ 
    return (a.val < b.val);
}

int main(int argc, char* argv[])
{
    auto start = high_resolution_clock::now();

    string data_image_path = argv[1];
    string query_image_path = argv[2];
    float threshold1 = stof(argv[3]); // for RMSD
    float threshold2 = stof(argv[4]); // for Gray-Scale image summary
    int n = stoi(argv[5]);
    int rows, cols;
    float imageSummaryQuery;
    

    // Read the data image
    ifstream data_image_file(data_image_path);
    data_image_file >> rows >> cols;
    data_image_file.close();
    float *data_imageV;
    int *data_imageR;
    int *data_imageG;
    int *data_imageB;
    float *compressedData;

    hipMallocManaged(&data_imageV, rows*cols*sizeof(float));
    hipMallocManaged(&data_imageR, rows*cols*sizeof(int));
    hipMallocManaged(&data_imageG, rows*cols*sizeof(int));
    hipMallocManaged(&data_imageB, rows*cols*sizeof(int));
    hipMallocManaged(&compressedData, rows*cols*sizeof(float));

    //cout<<"Going to read Data image"<<endl;
    //cout<<"Data file: "<<rows << "X" << cols<<endl;
    
    auto a1 = high_resolution_clock::now();
    dataInput(rows,cols,data_image_path,data_imageV,data_imageR,data_imageG,data_imageB,compressedData);
    auto a2 = high_resolution_clock::now();

    auto d1 = duration_cast<microseconds>(a2 - a1);
    //cout << "Time taken in reading Data file: "<< d1.count()/(1000000.0) << " sec" << endl;

    // Read the query image
    ifstream query_image_file(query_image_path);
    int query_rows, query_cols;
    query_image_file >> query_rows >> query_cols;
    query_image_file.close();
    float *query_imageV;
    int *query_imageR;
    int *query_imageG;
    int *query_imageB;

    hipMallocManaged(&query_imageV, query_rows*query_cols*sizeof(float));
    hipMallocManaged(&query_imageR, query_rows*query_cols*sizeof(int));
    hipMallocManaged(&query_imageG, query_rows*query_cols*sizeof(int));
    hipMallocManaged(&query_imageB, query_rows*query_cols*sizeof(int));
    
    //cout<<"Going to read Query image"<<endl;

    queryInput(query_rows,query_cols,query_image_path,query_imageV,query_imageR,query_imageG,query_imageB,imageSummaryQuery);

    //cout<<"Query Image Summary: "<<imageSummaryQuery<<endl;

    int imageSummarySize = (cols)*(rows)*3;

    float *imageSummary;
    float *rmsdValues;


    hipMallocManaged(&imageSummary, imageSummarySize*sizeof(float));
    hipMallocManaged(&rmsdValues, imageSummarySize*sizeof(float));


    int num_threads_per_block = 1024;
    int num_blocks = (rows*cols*3)/num_threads_per_block + 1;
    //cout<<"Number of blocks: "<<num_blocks<<endl;

    //cout<<"Going to enter the kernel"<<endl;

    computeImageSummary<<<num_blocks, 1024>>>(data_imageV,data_imageR, data_imageG, compressedData, data_imageB, query_imageV, query_imageR, query_imageG, query_imageB, rows, cols, query_rows, query_cols, imageSummary, rmsdValues, imageSummaryQuery,threshold2);

    hipError_t err = hipGetLastError();

     if ( err != hipSuccess )
     {
        printf("CUDA Error: %s\n", hipGetErrorString(err));       

        // Possibly: exit(-1) if program cannot continue....
     }


    hipDeviceSynchronize();

    //cout<<"Kernel job done !"<<endl;

    vector<triplet> output;

    for(int i=0;i<imageSummarySize;i++)
    {
        if(rmsdValues[i]!=-1 and rmsdValues[i] <= threshold1)
        {
            triplet t;
            t.x   = i/(cols*3);
            t.y   = (i%(cols*3))/3;
            t.orientation = (i%(cols*3))%3;
            t.val = rmsdValues[i];
            //t.summary = imageSummary[i];
            output.push_back(t);
            // cout<<t.val<<endl;
        }
    }
    sort(output.begin(),output.end(),sortbyVal);

    map<int,string> mp;
    mp[0] = "0";
    mp[1] = "45";
    mp[2] = "-45";

    fstream fsout;
    fsout.open("output.txt",ios::out);

    for(int i=0;i<min(n,(int)output.size());i++)
    {
        //cout << "x:" << output[i].x << ", y:" << output[i].y << ", orientation:" << mp[output[i].orientation] << ", RMSD:" << output[i].val <<", Image Summary:"<< output[i].summary - imageSummaryQuery<< "\n";
        fsout << output[i].x << " " << output[i].y << " " <<mp[output[i].orientation] <<endl;
    }

    //cout << "\nCount of Valid Candidates (after TH2): " << output.size()<<endl;

    fsout.close();

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    //cout << "Time taken by function: "<< duration.count()/(1000000.0) << " sec" << endl;


    // for(int i=0;i<imageSummarySize;i++){
    //     cout<<"x:"<<i/(cols*3)<<", y:"<<(i%(cols*3))/3<<", orientation:"<<(i%(cols*3))%3<<" -> "<<imageSummary[i]<<endl;
    // }


    return 0;

}